#include "hip/hip_runtime.h"
#include "CudaSmokeSim.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <chrono>
#include <iostream>
#include <memory>

int _initCounter = 0;

__forceinline__ __device__ int _IndexAt(int x, int y, int width) { return y * width + x; }
void _SwapPtr(float** l, float** r) { float* temp = *r; *r = *l; *l = temp; }

void _AddSource(int W, int H, float* x, float* s, float dt)
{
    int i, size = (W + 2) * (H + 2);
    for (i = 0; i < size; i++)
        x[i] += dt * s[i];
}

__global__ void SetBoundaryKernel(const int W, const int H, int b, float* x)
{
    for (int i = 1; i <= H; i++)
    {
        x[_IndexAt(0, i, W + 2)]        = b == 1 ? -x[_IndexAt(1, i, W + 2)] : x[_IndexAt(1, i, W + 2)];
        x[_IndexAt(W + 1, i, W + 2)]    = b == 1 ? -x[_IndexAt(W, i, W + 2)] : x[_IndexAt(W, i, W + 2)];
    }
    for (int i = 1; i <= W; i++)
    {
        x[_IndexAt(i, 0, W + 2)]        = b == 2 ? -x[_IndexAt(i, 1, W + 2)] : x[_IndexAt(i, 1, W + 2)];
        x[_IndexAt(i, H + 1, W + 2)]    = b == 2 ? -x[_IndexAt(i, H, W + 2)] : x[_IndexAt(i, H, W + 2)];
    }
    x[_IndexAt(0, 0, W + 2)]            = 0.5 * (x[_IndexAt(1, 0, W + 2)]       + x[_IndexAt(0, 1, W + 2)]);
    x[_IndexAt(0, H + 1, W + 2)]        = 0.5 * (x[_IndexAt(1, H + 1, W + 2)]   + x[_IndexAt(0, H, W + 2)]);
    x[_IndexAt(W + 1, 0, W + 2)]        = 0.5 * (x[_IndexAt(W, 0, W + 2)]       + x[_IndexAt(W + 1, 1, W + 2)]);
    x[_IndexAt(W + 1, H + 1, W + 2)]    = 0.5 * (x[_IndexAt(W, H + 1, W + 2)]   + x[_IndexAt(W + 1, H, W + 2)]);
}

__global__ void DiffuseKernel(float* x, const float* x0, const float a, const int BLOCK_COUNT, const int THREAD_COUNT, const int W, const int H)
{
    int startXIndex = 1 + (W * blockIdx.x) / BLOCK_COUNT;
    int endXIndex = 1 + (W * (blockIdx.x + 1)) / BLOCK_COUNT;
    int startYIndex = 1 + (H * threadIdx.x) / THREAD_COUNT;
    int endYIndex = 1 + (H * (threadIdx.x + 1)) / THREAD_COUNT;
    for (int k = 0; k < 4; k++)
    {
        for (int i = startXIndex; i < endXIndex; i++)
        {
            for (int j = startYIndex; j < endYIndex; j++)
            {
                int index = _IndexAt(i, j, W + 2);
                x[index] = (
                    x0[index] + a * (
                        x[index - 1] +
                        x[index + 1] +
                        x[index - (W + 2)] +
                        x[index + (W + 2)]
                    )
                ) / (1 + 4 * a);
            }
        }
    }
}

void _DiffuseVel(CudaSmokeSim_Context* ctx, float diff, float dt)
{
    float a = dt * diff;

    _SwapPtr(&ctx->dev_u_prev, &ctx->dev_u);
    _SwapPtr(&ctx->dev_v_prev, &ctx->dev_v);
    DiffuseKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_u, ctx->dev_u_prev, a, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    DiffuseKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_v, ctx->dev_v_prev, a, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 1, ctx->dev_u);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 2, ctx->dev_v);
}

void _DiffuseDens(CudaSmokeSim_Context* ctx, float diff, float dt)
{
    float a = dt * diff;

    _SwapPtr(&ctx->dev_dens_prev, &ctx->dev_dens);
    DiffuseKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_dens, ctx->dev_dens_prev, a, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
}

void _DiffuseTemp(CudaSmokeSim_Context* ctx, float diff, float dt)
{
    float a = dt * diff;

    _SwapPtr(&ctx->dev_temp_prev, &ctx->dev_temp);
    DiffuseKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_temp, ctx->dev_temp_prev, a, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
}

__global__ void AdvectKernel(float* d, const float* d0, const float* u, const float* v, const float dt0, const int BLOCK_COUNT, const int THREAD_COUNT, const int W, const int H)
{
    int startXIndex = 1 + (W * blockIdx.x) / BLOCK_COUNT;
    int endXIndex = 1 + (W * (blockIdx.x + 1)) / BLOCK_COUNT;
    int startYIndex = 1 + (H * threadIdx.x) / THREAD_COUNT;
    int endYIndex = 1 + (H * (threadIdx.x + 1)) / THREAD_COUNT;
    for (int i = startXIndex; i < endXIndex; i++)
    {
        for (int j = startYIndex; j < endYIndex; j++)
        {
            float x = i - dt0 * u[_IndexAt(i, j, W + 2)];
            float y = j - dt0 * v[_IndexAt(i, j, W + 2)];
            if (x < 0.5) x = 0.5;
            if (x > W + 0.5) x = W + 0.5;
            int i0 = (int)x;
            int i1 = i0 + 1;
            if (y < 0.5) y = 0.5;
            if (y > H + 0.5) y = H + 0.5;
            int j0 = (int)y;
            int j1 = j0 + 1;

            float s1 = x - i0;
            float s0 = 1 - s1;
            float t1 = y - j0;
            float t0 = 1 - t1;

            d[_IndexAt(i, j, W + 2)] =
                s0 * (t0 * d0[_IndexAt(i0, j0, W + 2)] + t1 * d0[_IndexAt(i0, j1, W + 2)]) +
                s1 * (t0 * d0[_IndexAt(i1, j0, W + 2)] + t1 * d0[_IndexAt(i1, j1, W + 2)]);
        }
    }
}

void _AdvectVel(CudaSmokeSim_Context* ctx, float dt)
{
    float dt0 = dt * ctx->height;
    _SwapPtr(&ctx->dev_u, &ctx->dev_u_prev);
    _SwapPtr(&ctx->dev_v, &ctx->dev_v_prev);
    AdvectKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_u, ctx->dev_u_prev, ctx->dev_u_prev, ctx->dev_v_prev, dt0, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    AdvectKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_v, ctx->dev_v_prev, ctx->dev_u_prev, ctx->dev_v_prev, dt0, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 1, ctx->dev_u);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 2, ctx->dev_v);

    //for (i = 1; i <= W; i++) {
    //    for (j = 1; j <= H; j++) {

    //        oldValSum += d0[_IndexAt(i, j)];
    //        newValSum += d[_IndexAt(i, j)];
    //    }
    //}
    //if (conserve && newValSum != 0.0f)
    //{
    //    float ratio = oldValSum / newValSum;
    //    for (int idx = 0; idx < W * H; idx++)
    //        d[idx] *= ratio;
    //}
}

void _AdvectDens(CudaSmokeSim_Context* ctx, float dt, bool conserve)
{
    float dt0 = dt * ctx->height;
    _SwapPtr(&ctx->dev_dens, &ctx->dev_dens_prev);
    AdvectKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_dens, ctx->dev_dens_prev, ctx->dev_u, ctx->dev_v, dt0, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 1, ctx->dev_dens);
}

void _AdvectTemp(CudaSmokeSim_Context* ctx, float dt, bool conserve)
{
    float dt0 = dt * ctx->height;
    _SwapPtr(&ctx->dev_temp, &ctx->dev_temp_prev);
    AdvectKernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_temp, ctx->dev_temp_prev, ctx->dev_u, ctx->dev_v, dt0, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 1, ctx->dev_temp);
}

__global__ void ProjectP1Kernel(const float* u, const float* v, float* p, float* div, const float h, const int BLOCK_COUNT, const int THREAD_COUNT, const int W, const int H)
{
    int startXIndex = 1 + (W * blockIdx.x) / BLOCK_COUNT;
    int endXIndex = 1 + (W * (blockIdx.x + 1)) / BLOCK_COUNT;
    int startYIndex = 1 + (H * threadIdx.x) / THREAD_COUNT;
    int endYIndex = 1 + (H * (threadIdx.x + 1)) / THREAD_COUNT;
    for (int i = startXIndex; i < endXIndex; i++)
    {
        for (int j = startYIndex; j < endYIndex; j++)
        {
            div[_IndexAt(i, j, W + 2)] = -0.5 * h * (
                u[_IndexAt(i + 1, j, W + 2)] - u[_IndexAt(i - 1, j, W + 2)] +
                v[_IndexAt(i, j + 1, W + 2)] - v[_IndexAt(i, j - 1, W + 2)]
            );
            p[_IndexAt(i, j, W + 2)] = 0;
        }
    }
}

__global__ void ProjectP2Kernel(float* p, const float* div, const int BLOCK_COUNT, const int THREAD_COUNT, const int W, const int H)
{
    int startXIndex = 1 + (W * blockIdx.x) / BLOCK_COUNT;
    int endXIndex = 1 + (W * (blockIdx.x + 1)) / BLOCK_COUNT;
    int startYIndex = 1 + (H * threadIdx.x) / THREAD_COUNT;
    int endYIndex = 1 + (H * (threadIdx.x + 1)) / THREAD_COUNT;
    for (int k = 0; k < 4; k++)
    {
        for (int i = startXIndex; i < endXIndex; i++)
        {
            for (int j = startYIndex; j < endYIndex; j++)
            {
                int index = _IndexAt(i, j, W + 2);
                p[index] = (
                    div[index] +
                    p[index - 1] +
                    p[index + 1] +
                    p[index - (W + 2)] +
                    p[index + (W + 2)]
                ) / 4;
            }
        }
    }
}

__global__ void ProjectP3Kernel(float* u, float* v, const float* p, const float h, const int BLOCK_COUNT, const int THREAD_COUNT, const int W, const int H)
{
    int startXIndex = 1 + (W * blockIdx.x) / BLOCK_COUNT;
    int endXIndex = 1 + (W * (blockIdx.x + 1)) / BLOCK_COUNT;
    int startYIndex = 1 + (H * threadIdx.x) / THREAD_COUNT;
    int endYIndex = 1 + (H * (threadIdx.x + 1)) / THREAD_COUNT;
    for (int i = startXIndex; i < endXIndex; i++)
    {
        for (int j = startYIndex; j < endYIndex; j++)
        {
            u[_IndexAt(i, j, W + 2)] -= 0.5 * (p[_IndexAt(i + 1, j, W + 2)] - p[_IndexAt(i - 1, j, W + 2)]) / h;
            v[_IndexAt(i, j, W + 2)] -= 0.5 * (p[_IndexAt(i, j + 1, W + 2)] - p[_IndexAt(i, j - 1, W + 2)]) / h;
        }
    }
}

void _ProjectVel(CudaSmokeSim_Context* ctx)
{
    float h = 1.0 / ctx->height;

    ProjectP1Kernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_u, ctx->dev_v, ctx->dev_u_prev, ctx->dev_v_prev, h, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 0, ctx->dev_u_prev);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 0, ctx->dev_v_prev);

    ProjectP2Kernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_u_prev, ctx->dev_v_prev, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 0, ctx->dev_u_prev);

    ProjectP3Kernel<<<ctx->blockCount, ctx->threadCount>>>(ctx->dev_u, ctx->dev_v, ctx->dev_u_prev, h, ctx->blockCount, ctx->threadCount, ctx->width, ctx->height);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 1, ctx->dev_u);
    SetBoundaryKernel<<<1, 1>>>(ctx->width, ctx->height, 2, ctx->dev_v);
}

CudaSmokeSim_Context* CudaSmokeSim_Init(int width, int height)
{
    if (_initCounter == 0)
    {
        hipError_t cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
            return nullptr;
        }
    }
    _initCounter++;

    std::unique_ptr<CudaSmokeSim_Context> ctx = std::make_unique<CudaSmokeSim_Context>();
    ctx->width = width;
    ctx->height = height;
    ctx->totalWidth = width + 2;
    ctx->totalHeight = height + 2;
    ctx->blockCount = 256;
    ctx->threadCount = 128;
    int size = ctx->totalWidth * ctx->totalHeight;

    if (hipMalloc((void**)&ctx->dev_u,         size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_v,         size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_u_prev,    size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_v_prev,    size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_dens,      size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_dens_prev, size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_temp,      size * sizeof(float)) != hipSuccess) return nullptr;
    if (hipMalloc((void**)&ctx->dev_temp_prev, size * sizeof(float)) != hipSuccess) return nullptr;

    return ctx.release();
}

void CudaSmokeSim_Uninit(CudaSmokeSim_Context* ctx)
{
    if (!ctx)
        return;

    hipFree(ctx->dev_u);
    hipFree(ctx->dev_v);
    hipFree(ctx->dev_u_prev);
    hipFree(ctx->dev_v_prev);
    hipFree(ctx->dev_dens);
    hipFree(ctx->dev_dens_prev);
    hipFree(ctx->dev_temp);
    hipFree(ctx->dev_temp_prev);

    _initCounter--;
    if (_initCounter == 0)
        hipDeviceReset();
}

void CudaSmokeSim_Step(CudaSmokeSim_Context* ctx, CudaSmokeSim_StepData* data)
{
    size_t arrSize = ctx->totalWidth * ctx->totalHeight * sizeof(float);

    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(ctx->dev_u,     data->u,    arrSize, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(ctx->dev_v,     data->v,    arrSize, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(ctx->dev_dens,  data->dens, arrSize, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(ctx->dev_temp,  data->temp, arrSize, hipMemcpyHostToDevice);

    // Velocity step
    if (data->velDiffusion > 0.0f)
        _DiffuseVel(ctx, data->velDiffusion, data->dt);
    _ProjectVel(ctx);
    _AdvectVel(ctx, data->dt);
    _ProjectVel(ctx);

    // Density step
    if (data->densDiffusion > 0.0f)
        _DiffuseDens(ctx, data->densDiffusion, data->dt);
    _AdvectDens(ctx, data->dt, true);

    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(data->u,    ctx->dev_u,     arrSize, hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(data->v,    ctx->dev_v,     arrSize, hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(data->dens, ctx->dev_dens,  arrSize, hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(data->temp, ctx->dev_temp,  arrSize, hipMemcpyDeviceToHost);
}